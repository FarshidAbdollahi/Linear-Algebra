#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "Utilities.cuh"

hipsparseHandle_t	handle;

hipsparseMatDescr_t	descrA = 0;
hipsparseMatDescr_t	descr_L = 0;
hipsparseMatDescr_t	descr_U = 0;

csrilu02Info_t		info_A = 0;
csrsv2Info_t		info_L = 0;
csrsv2Info_t		info_U = 0;

void				*pBuffer = 0;

/*****************************/
/* SETUP DESCRIPTOR FUNCTION */
/*****************************/
void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
	cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseSetMatType(descrA, matrixType));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrA, indexBase));
}

/**************************************************/
/* SETUP DESCRIPTOR FUNCTION FOR LU DECOMPOSITION */
/**************************************************/
void setUpDescriptorLU(hipsparseMatDescr_t &descrLU, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode, hipsparseDiagType_t diagType) {
	cusparseSafeCall(hipsparseCreateMatDescr(&descrLU));
	cusparseSafeCall(hipsparseSetMatType(descrLU, matrixType));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrLU, indexBase));
	cusparseSafeCall(hipsparseSetMatFillMode(descrLU, fillMode));
	cusparseSafeCall(hipsparseSetMatDiagType(descrLU, diagType));
}

/**********************************************/
/* MEMORY QUERY FUNCTION FOR LU DECOMPOSITION */
/**********************************************/
void memoryQueryLU(const int N, const int nnz, float * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, const hipsparseOperation_t matrixOperation) {

	cusparseSafeCall(hipsparseCreateCsrilu02Info(&info_A));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_L));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_U));

	int pBufferSize_M, pBufferSize_L, pBufferSize_U;
	cusparseSafeCall(hipsparseScsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M));
	cusparseSafeCall(hipsparseScsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L));
	cusparseSafeCall(hipsparseScsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U));

	int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
	gpuErrchk(hipMalloc(&pBuffer, pBufferSize));

}

void memoryQueryLU(const int N, const int nnz, double * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, const hipsparseOperation_t matrixOperation) {

	cusparseSafeCall(hipsparseCreateCsrilu02Info(&info_A));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_L));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_U));

	int pBufferSize_M, pBufferSize_L, pBufferSize_U;
	cusparseSafeCall(hipsparseDcsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M));
	cusparseSafeCall(hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L));
	cusparseSafeCall(hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U));

	int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
	gpuErrchk(hipMalloc(&pBuffer, pBufferSize));

}

/******************************************/
/* ANALYSIS FUNCTION FOR LU DECOMPOSITION */
/******************************************/
void analysisLUDecomposition(const int N, const int nnz, float * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2) {

	int structural_zero;

	cusparseSafeCall(hipsparseScsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer));
	hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

	cusparseSafeCall(hipsparseScsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer));
	cusparseSafeCall(hipsparseScsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer));

}

void analysisLUDecomposition(const int N, const int nnz, double * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2) {

	int structural_zero;

	cusparseSafeCall(hipsparseDcsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer));
	hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

	cusparseSafeCall(hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer));
	cusparseSafeCall(hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer));

}

/************************************************/
/* COMPUTE LU DECOMPOSITION FOR SPARSE MATRICES */
/************************************************/
void computeSparseLU(const int N, const int nnz, float * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy) {

	int numerical_zero;

	cusparseSafeCall(hipsparseScsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer));
	hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); }

}

void computeSparseLU(const int N, const int nnz, double * __restrict__ d_A, const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy) {

	int numerical_zero;

	cusparseSafeCall(hipsparseDcsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer));
	hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); }

}

/***********************************************************/
/* SOLVE SPARSE LINEAR SYSTEM BY LU DECOMPOSITION FUNCTION */
/***********************************************************/
void solveSparseLinearSystemLU(const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, float * __restrict__ d_A,
	                           const float * __restrict__ d_x, float * __restrict__ d_y, const int nnz, const int Nrows, hipsparseIndexBase_t indexBase)
{
	// --- Initialize cuSPARSE
	cusparseSafeCall(hipsparseCreate(&handle));

	const int Ncols = Nrows;                    // --- Number of columns
	const int N		= Nrows;
	const int lda	= Nrows;                    // --- Leading dimension of dense matrix

	/*********************************************/
	/* STEP 1: CREATE DESCRIPTORS FOR A, L AND U */
	/*********************************************/
	setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase);
	setUpDescriptorLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
	setUpDescriptorLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	/**************************************************************************************************/
	/* STEP 2: QUERY HOW MUCH MEMORY USED IN LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/**************************************************************************************************/
	memoryQueryLU(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	/************************************************************************************************/
	/* STEP 3: ANALYZE THE THREE PROBLEMS: LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/************************************************************************************************/
	analysisLUDecomposition(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, HIPSPARSE_SOLVE_POLICY_USE_LEVEL);

	/************************************/
	/* STEP 4: FACTORIZATION: A = L * U */
	/************************************/
	computeSparseLU(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_SOLVE_POLICY_NO_LEVEL);

	/*********************/
	/* STEP 5: L * z = x */
	/*********************/
	// --- Allocating the intermediate result vector
	float *d_z;        gpuErrchk(hipMalloc(&d_z, N * sizeof(float)));

	const float alpha = 1.;
	cusparseSafeCall(hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));

	/*********************/
	/* STEP 5: U * y = z */
	/*********************/
	cusparseSafeCall(hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));

}

void solveSparseLinearSystemLU(const int * __restrict__ d_A_RowIndices, const int * __restrict__ d_A_ColIndices, double * __restrict__ d_A,
							   const double * __restrict__ d_x, double * __restrict__ d_y, const int nnz, const int Nrows, hipsparseIndexBase_t indexBase)
{
	// --- Initialize cuSPARSE
	cusparseSafeCall(hipsparseCreate(&handle));

	const int Ncols = Nrows;                    // --- Number of columns
	const int N = Nrows;
	const int lda = Nrows;                    // --- Leading dimension of dense matrix

	/*********************************************/
	/* STEP 1: CREATE DESCRIPTORS FOR A, L AND U */
	/*********************************************/
	setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase);
	setUpDescriptorLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
	setUpDescriptorLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, indexBase, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	/**************************************************************************************************/
	/* STEP 2: QUERY HOW MUCH MEMORY USED IN LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/**************************************************************************************************/
	memoryQueryLU(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	/************************************************************************************************/
	/* STEP 3: ANALYZE THE THREE PROBLEMS: LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/************************************************************************************************/
	analysisLUDecomposition(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, HIPSPARSE_SOLVE_POLICY_USE_LEVEL);

	/************************************/
	/* STEP 4: FACTORIZATION: A = L * U */
	/************************************/
	computeSparseLU(N, nnz, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_SOLVE_POLICY_NO_LEVEL);

	/*********************/
	/* STEP 5: L * z = x */
	/*********************/
	// --- Allocating the intermediate result vector
	double *d_z;        gpuErrchk(hipMalloc(&d_z, N * sizeof(double)));

	const double alpha = 1.;
	cusparseSafeCall(hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));

	/*********************/
	/* STEP 5: U * y = z */
	/*********************/
	cusparseSafeCall(hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));

}
