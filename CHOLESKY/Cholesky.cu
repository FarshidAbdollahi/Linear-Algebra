#include "hip/hip_runtime.h"
#include "device_launch_paraMeters.h"

#include<iostream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"

/********/
/* MAIN */
/********/
int main(){

	const int Nrows = 5;
	const int Ncols = 5;

	// --- Setting the host, Nrows x Ncols matrix
	double h_A[Nrows][Ncols] = { 
		{ 1.,    -1.,    -1.,    -1.,    -1.,},  
		{-1.,     2.,     0.,     0.,     0.,}, 
		{-1.,     0.,     3.,     1.,     1.,}, 
		{-1.,     0.,     1.,     4.,     2.,}, 
		{-1.,     0.,     1.,     2.,     5.,}
	};
	
	printf("Original matrix\n");
	for(int i = 0; i < Nrows; i++)
		for(int j = 0; j < Ncols; j++)
			printf("L[%i, %i] = %f\n", i, j, h_A[i][j]);

	// --- Setting the device matrix and moving the host matrix to the device
	double *d_A;			gpuErrchk(hipMalloc(&d_A,		Nrows * Ncols * sizeof(double)));
	gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

	// --- cuSOLVE input/output parameters/arrays
	int work_size = 0;
	int *devInfo;			gpuErrchk(hipMalloc(&devInfo,	        sizeof(int)));
	
	// --- CUDA solver initialization
	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);

	// --- CUDA CHOLESKY initialization
	cusolveSafeCall(hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, &work_size));

	// --- CUDA POTRF execution
	double *work;	gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));
	cusolveSafeCall(hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, work, work_size, devInfo));
	int devInfo_h = 0;	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	if (devInfo_h != 0) std::cout	<< "Unsuccessful potrf execution\n\n";

	// --- At this point, the upper triangular part of A contains the elements of L. Showing this.
	printf("\nFactorized matrix\n");
	gpuErrchk(hipMemcpy(h_A, d_A, Nrows * Ncols * sizeof(double), hipMemcpyDeviceToHost));
	for(int i = 0; i < Nrows; i++)
		for(int j = 0; j < Ncols; j++)
			if (i <= j) printf("L[%i, %i] = %f\n", i, j, h_A[i][j]);

	hipsolverDnDestroy(solver_handle);

	return 0;

}
