#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

//#define FULLSVD
//#define PRINTRESULTS

/********/
/* MAIN */
/********/
int main() {
	
	const int			M			= 3;
	const int			N			= 3;
	const int			lda			= M;
	//const int			numMatrices = 3;
	const int			numMatrices = 16384;

	TimingGPU timerGPU;
	
	// --- Setting the host matrix
	double *h_A = (double *)malloc(lda * N * numMatrices * sizeof(double));
	for (unsigned int k = 0; k < numMatrices; k++)
		for (unsigned int i = 0; i < M; i++){
			for (unsigned int j = 0; j < N; j++){
				h_A[k * M * N + j * M + i] = (1. / (k + 1)) * (i + j * j) * (i + j);
				//printf("%d %d %f\n", i, j, h_A[j*M + i]);
			}
		}
	
	// --- Setting the device matrix and moving the host matrix to the device
	double *d_A;         gpuErrchk(hipMalloc(&d_A, M * N * numMatrices * sizeof(double)));
	gpuErrchk(hipMemcpy(d_A, h_A, M * N * numMatrices * sizeof(double), hipMemcpyHostToDevice));
	
	// --- host side SVD results space
	double *h_S = (double *)malloc(N *     numMatrices * sizeof(double));
	double *h_U = NULL;
	double *h_V = NULL;
#ifdef FULLSVD
			h_U = (double *)malloc(M * M * numMatrices * sizeof(double));
			h_V = (double *)malloc(N * N * numMatrices * sizeof(double));
#endif

	// --- device side SVD workspace and matrices
	int work_size = 0;

	int	*devInfo;		 gpuErrchk(hipMalloc(&devInfo, sizeof(int)));
	double *d_S;         gpuErrchk(hipMalloc(&d_S, N *     numMatrices * sizeof(double)));
	double *d_U = NULL;
	double *d_V = NULL;
#ifdef FULLSVD
						 gpuErrchk(hipMalloc(&d_U, M * M * numMatrices * sizeof(double)));
						 gpuErrchk(hipMalloc(&d_V, N * N * numMatrices * sizeof(double)));
#endif

	double *d_work = NULL; /* devie workspace for gesvdj */
	int devInfo_h = 0; /* host copy of error devInfo_h */

	// --- Parameters configuration of Jacobi-based SVD
	const double			tol				= 1.e-7;
	const int				maxSweeps		= 15;
		  hipsolverEigMode_t jobz;									// --- HIPSOLVER_EIG_MODE_VECTOR - Compute eigenvectors; HIPSOLVER_EIG_MODE_NOVECTOR - Compute singular values only
#ifdef FULLSVD
		jobz = HIPSOLVER_EIG_MODE_VECTOR;
#else
		jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
#endif

	const int				econ			= 0;							// --- econ = 1 for economy size 
	
	// --- Numerical result parameters of gesvdj 
	double					residual		= 0;
	int						executedSweeps	= 0;
	
	// --- CUDA solver initialization
	hipsolverHandle_t solver_handle = NULL;
	cusolveSafeCall(hipsolverDnCreate(&solver_handle));

	// --- Configuration of gesvdj
	hipsolverGesvdjInfo_t gesvdj_params = NULL;
	cusolveSafeCall(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

	// --- Set the computation tolerance, since the default tolerance is machine precision
	cusolveSafeCall(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));

	// --- Set the maximum number of sweeps, since the default value of max. sweeps is 100
	cusolveSafeCall(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, maxSweeps));

	// --- Query the SVD workspace 
	cusolveSafeCall(hipsolverDnDgesvdjBatched_bufferSize(
		solver_handle,
		jobz,										// --- Compute the singular vectors or not
		M,											// --- Nubmer of rows of A, 0 <= M
		N,											// --- Number of columns of A, 0 <= N 
		d_A,										// --- M x N
		lda,										// --- Leading dimension of A
		d_S,										// --- Square matrix of size min(M, N) x min(M, N)
		d_U,										// --- M x M if econ = 0, M x min(M, N) if econ = 1
		lda,										// --- Leading dimension of U, ldu >= max(1, M)
		d_V,										// --- N x N if econ = 0, N x min(M,N) if econ = 1
		lda,										// --- Leading dimension of V, ldv >= max(1, N)
		&work_size,
		gesvdj_params,
		numMatrices));

	gpuErrchk(hipMalloc(&d_work, sizeof(double) * work_size));

	// --- Compute SVD
	timerGPU.StartCounter();
	cusolveSafeCall(hipsolverDnDgesvdjBatched(
		solver_handle,
		jobz,										// --- Compute the singular vectors or not
		M,											// --- Number of rows of A, 0 <= M
		N,											// --- Number of columns of A, 0 <= N 
		d_A,										// --- M x N
		lda,										// --- Leading dimension of A
		d_S,										// --- Square matrix of size min(M, N) x min(M, N)
		d_U,										// --- M x M if econ = 0, M x min(M, N) if econ = 1
		lda,										// --- Leading dimension of U, ldu >= max(1, M)
		d_V,										// --- N x N if econ = 0, N x min(M, N) if econ = 1
		lda,										// --- Leading dimension of V, ldv >= max(1, N)
		d_work,
		work_size,
		devInfo,
		gesvdj_params,
		numMatrices));

	printf("Calculation of the singular values only: %f ms\n\n", timerGPU.GetCounter());

	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_S, d_S, sizeof(double) *       N * numMatrices, hipMemcpyDeviceToHost));
#ifdef FULLSVD
	gpuErrchk(hipMemcpy(h_U, d_U, sizeof(double) * lda * M * numMatrices, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_V, d_V, sizeof(double) * lda * N * numMatrices, hipMemcpyDeviceToHost));
#endif

#ifdef PRINTRESULTS
	printf("SINGULAR VALUES \n");
	printf("_______________ \n");
	for (int k = 0; k < numMatrices; k++) {
		for (int p = 0; p < N; p++)
			printf("Matrix nr. %d; SV nr. %d; Value = %f\n", k, p, h_S[k * N + p]);
		printf("\n");
	}
#ifdef FULLSVD
	printf("SINGULAR VECTORS U \n");
	printf("__________________ \n");
	for (int k = 0; k < numMatrices; k++) {
		for (int q = 0; q < (1 - econ) * M + econ * min(M, N); q++)
			for (int p = 0; p < M; p++)
				printf("Matrix nr. %d; U nr. %d; Value = %f\n", k, p, h_U[((1 - econ) * M + econ * min(M, N)) * M * k + q * M + p]);
		printf("\n");
	}

	printf("SINGULAR VECTORS V \n");
	printf("__________________ \n");
	for (int k = 0; k < numMatrices; k++) {
		for (int q = 0; q < (1 - econ) * N + econ * min(M, N); q++)
			for (int p = 0; p < N; p++)
				printf("Matrix nr. %d; V nr. %d; Value = %f\n", k, p, h_V[((1 - econ) * N + econ * min(M, N)) * N * k + q * N + p]);
		printf("\n");
	}
#endif
#endif

	if (0 == devInfo_h){
		printf("gesvdj converges \n");
	}
	else if (0 > devInfo_h){
		printf("%d-th parameter is wrong \n", -devInfo_h);
		exit(1);
	}
	else{
		printf("WARNING: devInfo_h = %d : gesvdj does not converge \n", devInfo_h);
	}

	// --- Free resources
	if (d_A) gpuErrchk(hipFree(d_A));
	if (d_S) gpuErrchk(hipFree(d_S));
#ifdef FULLSVD
	if (d_U) gpuErrchk(hipFree(d_U));
	if (d_V) gpuErrchk(hipFree(d_V));
#endif
	if (devInfo) gpuErrchk(hipFree(devInfo));
	if (d_work) gpuErrchk(hipFree(d_work));
	if (solver_handle) cusolveSafeCall(hipsolverDnDestroy(solver_handle));
	if (gesvdj_params) cusolveSafeCall(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
	
	gpuErrchk(hipDeviceReset());
	
	return 0;
}
