#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "Utilities.cuh"

/********/
/* MAIN */
/********/
int main()
{
	// --- Initialize cuSPARSE
	hipsparseHandle_t handle;	cusparseSafeCall(hipsparseCreate(&handle));

	const int N     = 5;		// --- Size of the linear system

	// --- Lower diagonal, diagonal and upper diagonal of the system matrix
	double *h_ld = (double*)malloc(N * sizeof(double));
	double *h_d  = (double*)malloc(N * sizeof(double));
	double *h_ud = (double*)malloc(N * sizeof(double));
	
	h_ld[0]		= 0.;
	h_ud[N-1]	= 0.;
	for (int k = 0; k < N - 1; k++) {
		h_ld[k + 1] = -1.;
		h_ud[k]     = -1.;
	}

	for (int k = 0; k < N; k++) h_d[k] = 2.;

	double *d_ld;	gpuErrchk(hipMalloc(&d_ld, N * sizeof(double)));
	double *d_d;	gpuErrchk(hipMalloc(&d_d,  N * sizeof(double)));
	double *d_ud;	gpuErrchk(hipMalloc(&d_ud, N * sizeof(double)));
	
	gpuErrchk(hipMemcpy(d_ld, h_ld, N * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_d,  h_d,  N * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_ud, h_ud, N * sizeof(double), hipMemcpyHostToDevice));
	
    // --- Allocating and defining dense host and device data vectors
	double *h_x	= (double *)malloc(N * sizeof(double)); 
	h_x[0] = 100.0;  h_x[1] = 200.0; h_x[2] = 400.0; h_x[3] = 500.0; h_x[4] = 300.0;

	double *d_x;		gpuErrchk(hipMalloc(&d_x, N * sizeof(double)));   
    gpuErrchk(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
	
	// --- Allocating the host and device side result vector
	double *h_y	= (double *)malloc(N * sizeof(double)); 
	double *d_y;		gpuErrchk(hipMalloc(&d_y, N * sizeof(double))); 

	cusparseSafeCall(cusparseDgtsv(handle, N, 1, d_ld, d_d, d_ud, d_x, N));

	hipMemcpy(h_x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
	for (int k=0; k<N; k++) printf("%f\n", h_x[k]);
}
